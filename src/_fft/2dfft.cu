#include<stdio.h>
#include<stdlib.h>
#include<complex.h>
#include<hip/hip_runtime.h>
#include<hip/hip_runtime_api.h>
#include<hipfft/hipfft.h>
#include"comfft.h"

int fft2d(float complex *src_data, float complex *dst_data, int nx, int ny){
    hipfftHandle plan;    
    hipfftComplex *dev_src, *dev_dst;
    //Allocate memory on the GPU and copy over the src array
    cudaSafeCall(hipMalloc((void**) &dev_src, sizeof(hipfftComplex)*nx*ny));
    cudaSafeCall(hipMalloc((void**) &dev_dst, sizeof(hipfftComplex)*nx*ny));
    cudaSafeCall(hipMemcpy(dev_src, src_data, sizeof(hipfftComplex)*nx*ny, hipMemcpyHostToDevice));
    //Create a 2d fft plan
    //cufft functions return cufftResults, which require different error handling
    cufftSafeCall(hipfftPlan2d(&plan, nx, ny, HIPFFT_C2C));
    cufftSafeCall(hipfftExecC2C(plan, dev_src, dev_dst, HIPFFT_FORWARD));
    cudaSafeCall(hipMemcpy(dst_data, dev_dst, sizeof(hipfftComplex)*nx*ny, hipMemcpyDeviceToHost));
    CudaCheckError();    
    //Free GPU memory
    cufftSafeCall(hipfftDestroy(plan));
    hipFree(dev_src);
    hipFree(dev_dst);
    return 0;
}
