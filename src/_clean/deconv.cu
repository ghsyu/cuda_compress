#include "hip/hip_runtime.h"
/*
 * Some additional deconvolution functions for AIPY, written in C++ and CUDA.  These are
 * mostly for speed-critical applications. 
 *
 * Author: Aaron Parsons, Gilbert Hsyu
 */

#include "deconv.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>


__global__ void clean2dc(int *dim1_p, int *dim2_p, int *argmax1_p, int *argmax2_p, float *stepr_p, \
                        float *stepi_p, float *ker, float *res, float *val_arr){ 
    int dim1 = *dim1_p;
    int dim2 = *dim2_p;
    int argmax1 = *argmax1_p;
    int argmax2 = *argmax2_p;
    float stepr = *stepr_p;
    float stepi = *stepi_p;
    float valr, vali;
    int gridx = (dim1 % BLOCKSIZEX == 0) ? dim1/BLOCKSIZEX : dim1/BLOCKSIZEX + 1;
    int n1 = threadIdx.x + blockIdx.x * blockDim.x;
    int n2 = threadIdx.y + blockIdx.y * blockDim.y;
    if ((n1 < dim1) && (n2 < dim2)){
        int wrap_n1 = (n1 + argmax1) % dim1;
        int wrap_n2 = (n2 + argmax2) % dim2;
        res[2*(wrap_n1 + wrap_n2*dim1)]     -= (ker[2*(n1 + n2*dim1)] * stepr - ker[2*(n1 + n2*dim1)+1] * stepi);
        res[2*(wrap_n1 + wrap_n2*dim1) + 1] -= (ker[2*(n1 + n2*dim1)] * stepi + ker[2*(n1 + n2*dim1)+1] * stepr);
        valr = res[2*(wrap_n1 + wrap_n2*dim1)];
        vali = res[2*(wrap_n1 + wrap_n2*dim1) + 1];
        val_arr[2*(n1 + blockDim.x*gridx*n2)]     = valr;
        val_arr[2*(n1 + blockDim.x*gridx*n2) + 1] = vali;
    }
    return;
}

//   ____ _                  ____     _      
//  / ___| | ___  __ _ _ __ |___ \ __| | ___ 
// | |   | |/ _ \/ _` | '_ \  __) / _` |/ __|
// | |___| |  __/ (_| | | | |/ __/ (_| | (__ 
//  \____|_|\___|\__,_|_| |_|_____\__,_|\___|  
// Does a 2d complex-valued clean
float *clean_2d_c_GPU(float *res, float *ker,
        double gain, int maxiter, \
        int argmax1, int argmax2, \
        int stop_if_div, \
        float *stepr_p, float *stepi_p, \
        int ker_len, int res_len, int dim1, int dim2, float* retval) {
    int gridx, gridy;
    float *dev_ker, *dev_res, *dev_stepr, *dev_stepi, *dev_val_arr;
    int *dev_argmax1, *dev_argmax2,
        *dev_dim1, *dev_dim2;
    float stepr = *stepr_p;
    float stepi = *stepi_p;
    //Ceiling division of dim1/BLOCKSIZEX and dim2/BLOCKSIZEY
    gridx = (dim1 % BLOCKSIZEX == 0) ? dim1/BLOCKSIZEX : dim1/BLOCKSIZEX + 1;
    gridy = (dim2 % BLOCKSIZEY == 0) ? dim2/BLOCKSIZEY : dim2/BLOCKSIZEY + 1;
    dim3 grid(gridx, gridy);
    dim3 blocksize(BLOCKSIZEX, BLOCKSIZEY);
    //arr_size will be different from res_len if a dimension of res is not a multiple of 16.
    int arr_size = BLOCKSIZEX*BLOCKSIZEY*gridx*gridy;
    float *val_arr;
    val_arr = (float *)malloc(2*arr_size*sizeof(float));
    if (val_arr == NULL){
        exit (EXIT_FAILURE);
    }
    
    CudaSafeCall(hipMalloc((void**) &dev_ker,      ker_len));
    CudaSafeCall(hipMalloc((void**) &dev_res,      res_len));
    CudaSafeCall(hipMalloc((void**) &dev_dim1,     sizeof(int)));
    CudaSafeCall(hipMalloc((void**) &dev_dim2,     sizeof(int)));
    CudaSafeCall(hipMalloc((void**) &dev_argmax1,  sizeof(int)));
    CudaSafeCall(hipMalloc((void**) &dev_argmax2,  sizeof(int)));
    CudaSafeCall(hipMalloc((void**) &dev_stepr,    sizeof(float)));
    CudaSafeCall(hipMalloc((void**) &dev_stepi,    sizeof(float)));
    CudaSafeCall(hipMalloc((void**) &dev_val_arr,  2*sizeof(float)*(gridx*BLOCKSIZEX*gridy*BLOCKSIZEY)));
    
    CudaSafeCall(hipMemcpy(dev_ker,      ker,      ker_len,       hipMemcpyHostToDevice));
    CudaSafeCall(hipMemcpy(dev_res,      res,      res_len,       hipMemcpyHostToDevice));
    CudaSafeCall(hipMemcpy(dev_dim1,     &dim1,    sizeof(int),   hipMemcpyHostToDevice));
    CudaSafeCall(hipMemcpy(dev_dim2,     &dim2,    sizeof(int),   hipMemcpyHostToDevice));
    CudaSafeCall(hipMemcpy(dev_argmax1,  &argmax1, sizeof(int),   hipMemcpyHostToDevice));
    CudaSafeCall(hipMemcpy(dev_argmax2,  &argmax2, sizeof(int),   hipMemcpyHostToDevice));
    CudaSafeCall(hipMemcpy(dev_stepr,    &stepr,   sizeof(float), hipMemcpyHostToDevice));
    CudaSafeCall(hipMemcpy(dev_stepi,    &stepi,   sizeof(float), hipMemcpyHostToDevice));

    // Take next step and compute score
    clean2dc<<<grid, blocksize>>>(dev_dim1, dev_dim2, dev_argmax1, dev_argmax2, dev_stepr, \
                                dev_stepi, dev_ker, dev_res, dev_val_arr);
    hipDeviceSynchronize();
    CudaCheckError();
    CudaSafeCall(hipMemcpy(val_arr, dev_val_arr, 2*sizeof(float)*arr_size,  hipMemcpyDeviceToHost));
    CudaSafeCall(hipMemcpy(res,     dev_res,     res_len,                   hipMemcpyDeviceToHost));
    hipFree(dev_ker);
    hipFree(dev_res);
    hipFree(dev_dim1);
    hipFree(dev_dim2);
    hipFree(dev_argmax1);
    hipFree(dev_argmax2);
    hipFree(dev_stepr);
    hipFree(dev_stepi);
    hipFree(dev_val_arr);
    for(int i = 0, j = 0; i < 2*arr_size; i++){
        if(i % (2*BLOCKSIZEX) < 2*dim1 && i/(2*BLOCKSIZEX) < 2*dim2){
            retval[j] = val_arr[i];
            j++;
        }
    }
    free(val_arr);
    return retval;
}