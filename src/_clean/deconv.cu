#include "hip/hip_runtime.h"
/*
 * Some additional deconvolution functions for AIPY, written in C++ and CUDA.  These are
 * mostly for speed-critical applications. 
 *
 * Author: Aaron Parsons, Gilbert Hsyu
 */

#include "deconv.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <stdlib.h>
#include <stdio.h>
//XXX The area matrix is currently not working
__global__ void sum_max(unsigned int n, int * area, float *g_nscore_i, float *g_max_i, int* g_max_idx_i, \
                        float *g_nscore_o, float *g_max_o, int *g_max_idx_o){
    extern __shared__ float smem[];
    float *s_nscore = smem;
    float *s_max    = &s_nscore[blockDim.x];
    int *s_max_idx  = (int *) &s_max[2*blockDim.x];
    int tid = threadIdx.x;
    int i = blockIdx.x*blockDim.x + tid;
    if (i < n ){
        s_nscore[tid]      = g_nscore_i[i];
        s_max[2*tid]       = g_max_i[2*i];
        s_max[2*tid+1]     = g_max_i[2*i+1];
        s_max_idx[2*tid]   = g_max_idx_i[2*i];
        s_max_idx[2*tid+1] = g_max_idx_i[2*i+1];
    } else {
        s_nscore[tid]      = 0;
        s_max[2*tid]       = 0;
        s_max[2*tid+1]     = 0;
        s_max_idx[2*tid]   = 0;
        s_max_idx[2*tid+1] = 0;
    }
    __syncthreads();
    for (unsigned int s=blockDim.x/2; s>0; s>>=1){
        if (tid < s){
            s_nscore[tid] += s_nscore[tid + s];
            if ((s_max[2*tid]*s_max[2*tid] + s_max[2*tid+1]*s_max[2*tid+1]) < \
                (s_max[2*(tid+s)]*s_max[2*(tid+s)] + s_max[2*(tid+s)+1]*s_max[2*(tid+s)+1])){
                printf("Before: %f (%d, %d), %f (%d, %d)\n", s_max[2*tid], s_max_idx[2*tid], s_max_idx[2*tid+1], s_max[2*(tid+s)], s_max_idx[2*(tid+s)], s_max_idx[2*(tid+s)+1]);
                s_max[2*tid]         = s_max[2*(tid+s)];
                s_max[2*tid+1]       = s_max[2*(tid+s)+1];
                s_max_idx[2*tid]     = s_max_idx[2*(tid+s)];
                s_max_idx[2*tid+1]   = s_max_idx[2*(tid+s)+1];
                printf("After: %f (%d, %d)\n", s_max[2*tid], s_max_idx[2*tid], s_max_idx[2*tid+1]);
            }
        }
        __syncthreads();
    }
    if (tid == 0){
        printf("smax: %f ", s_max[0]);
        printf("s_max_idx: (%d, %d)\n", s_max_idx[0], s_max_idx[1]);
        g_nscore_o[blockIdx.x]        = s_nscore[0];
        g_max_o[2*blockIdx.x]         = s_max[0];
        g_max_o[2*blockIdx.x+1]       = s_max[1];
        g_max_idx_o[2*blockIdx.x]     = s_max_idx[0];
        g_max_idx_o[2*blockIdx.x+1]   = s_max_idx[1];
    }
}

__global__ void clean2dc(unsigned int dim1, unsigned int dim2, unsigned int argmax1, unsigned int argmax2, float stepr, \
                        float stepi, float *ker, float *res, float *g_nscore, \
                        float *g_max, int *g_max_idx){
    float valr, vali;
    int n1 = threadIdx.x + blockIdx.x * blockDim.x;
    int n2 = threadIdx.y + blockIdx.y * blockDim.y;
    int i = n1 + n2 * dim1;
    if ((n1 < dim1) && (n2 < dim2)){
        int wrap_n1 = (n1 + argmax1) % dim1;
        int wrap_n2 = (n2 + argmax2) % dim2;
        if (ker[2*i] == 1){
            printf("argmax: (%d, %d), idx: %d, stepr: %f, res: %f,\n", argmax1, argmax2, wrap_n1+wrap_n2*dim1, stepr, res[2*(wrap_n1+wrap_n2*dim1)]);
        }
        res[2*(wrap_n1 + wrap_n2*dim1)]     -= (ker[2*(i)] * stepr - ker[2*(i)+1] * stepi);
        res[2*(wrap_n1 + wrap_n2*dim1) + 1] -= (ker[2*(i)] * stepi + ker[2*(i)+1] * stepr);
        
        if (ker[2*i] == 1){
            printf("res: %f\n", res[2*(wrap_n1+wrap_n2*dim1)]);
        }
        valr = res[2*(wrap_n1 + wrap_n2*dim1)];
        vali = res[2*(wrap_n1 + wrap_n2*dim1) + 1];
        g_nscore[i] = valr*valr+vali*vali;
        g_max[2*i] = valr;
        g_max[2*i+1] = vali;
        g_max_idx[2*i] = n1;
        g_max_idx[2*i+1] = n2;
    }
    return;
}

//   ____ _                  ____     _      
//  / ___| | ___  __ _ _ __ |___ \ __| | ___ 
// | |   | |/ _ \/ _` | '_ \  __) / _` |/ __|
// | |___| |  __/ (_| | | | |/ __/ (_| | (__ 
//  \____|_|\___|\__,_|_| |_|_____\__,_|\___|  
// Does a 2d complex-valued clean
float *clean_2d_c_GPU(float *res, float *ker, int64_t * area, \
        double gain, int maxiter, \
        int stop_if_div, \
        float stepr, float stepi, int argmax1, int argmax2, \
        int ker_len, int res_len, int area_len, int dim1, int dim2,
        float *nscore_p, float *maxr_p, float *maxi_p, int *nargmax1_p, int *nargmax2_p) {
    int gridx, gridy;
    float *dev_ker, *dev_res, *g_nscore_i, *g_max_i, *g_nscore_o, *g_max_o;
    int *dev_area, *g_max_idx_i, *g_max_idx_o;
    float max_p[2];
    int max_idx_p[2];
    int gridsize;
    //Ceiling division of dim1/BLOCKSIZEX and dim2/BLOCKSIZEY
    gridx = (dim1 % BLOCKSIZEX == 0) ? dim1/BLOCKSIZEX : dim1/BLOCKSIZEX + 1;
    gridy = (dim2 % BLOCKSIZEY == 0) ? dim2/BLOCKSIZEY : dim2/BLOCKSIZEY + 1;
    dim3 grid(gridx, gridy);
    dim3 blocksize(BLOCKSIZEX, BLOCKSIZEY);
    CudaSafeCall(hipMalloc((void**) &dev_ker,      ker_len));
    CudaSafeCall(hipMalloc((void**) &dev_res,      res_len));
    CudaSafeCall(hipMalloc((void**) &dev_area,     area_len));
    CudaSafeCall(hipMalloc((void**) &g_nscore_i,   sizeof(float)*dim1*dim2));
    CudaSafeCall(hipMalloc((void**) &g_max_i,      2*sizeof(float)*dim1*dim2));
    CudaSafeCall(hipMalloc((void**) &g_max_idx_i,  2*sizeof(int)*dim1*dim2));
    CudaSafeCall(hipMalloc((void**) &g_nscore_o,   sizeof(float)*dim1*dim2));
    CudaSafeCall(hipMalloc((void**) &g_max_o,      2*sizeof(float)*dim1*dim2));
    CudaSafeCall(hipMalloc((void**) &g_max_idx_o,  2*sizeof(int)*dim1*dim2));    
    CudaSafeCall(hipMemcpy(dev_ker,      ker,      ker_len,       hipMemcpyHostToDevice));
    CudaSafeCall(hipMemcpy(dev_res,      res,      res_len,       hipMemcpyHostToDevice));
    CudaSafeCall(hipMemcpy(dev_area,     area,     area_len,      hipMemcpyHostToDevice));
    
    // Take next step and compute score
    clean2dc<<<grid, blocksize>>>(dim1, dim2, argmax1, argmax2, stepr, \
                                stepi, dev_ker, dev_res, g_nscore_i, g_max_i, g_max_idx_i);
    CudaSafeCall(hipMemcpy(res, dev_res, res_len, hipMemcpyDeviceToHost));
    CudaCheckError();
    //Make the kernel invocation 1D
    int bsize = BLOCKSIZEX*BLOCKSIZEY;
    int smemsize = 3*BLOCKSIZEX*BLOCKSIZEY*sizeof(float)+2*BLOCKSIZEX*BLOCKSIZEY*sizeof(int);
    int len = dim1*dim2;
    while(len > bsize){
        gridsize = (len % bsize == 0) ? len/bsize : len/bsize + 1;
        sum_max<<<gridsize, bsize, smemsize>>>(len, dev_area, g_nscore_i, g_max_i, g_max_idx_i, g_nscore_o, g_max_o, g_max_idx_o);
        CudaCheckError();
        len = gridsize;
        CudaSafeCall(hipMemcpy(g_nscore_i, g_nscore_o,   sizeof(float)*dim1*dim2,   hipMemcpyDeviceToDevice));
        CudaSafeCall(hipMemcpy(g_max_i, g_max_o,         2*sizeof(float)*dim1*dim2, hipMemcpyDeviceToDevice));
        CudaSafeCall(hipMemcpy(g_max_idx_i, g_max_idx_o, 2*sizeof(int)*dim1*dim2,   hipMemcpyDeviceToDevice));
    }

    gridsize = (len % bsize == 0) ? len/bsize : len/bsize + 1;
    sum_max<<<gridsize, bsize, smemsize>>>(len, dev_area, g_nscore_i, g_max_i, g_max_idx_i, g_nscore_o, g_max_o, g_max_idx_o);
    CudaCheckError();
    len = gridsize;
    CudaSafeCall(hipMemcpy(g_nscore_i, g_nscore_o,   sizeof(float)*dim1*dim2,   hipMemcpyDeviceToDevice));
    CudaSafeCall(hipMemcpy(g_max_i, g_max_o,         2*sizeof(float)*dim1*dim2, hipMemcpyDeviceToDevice));
    CudaSafeCall(hipMemcpy(g_max_idx_i, g_max_idx_o, 2*sizeof(int)*dim1*dim2,   hipMemcpyDeviceToDevice));

    CudaCheckError();
    CudaSafeCall(hipMemcpy(nscore_p, g_nscore_o, sizeof(float), hipMemcpyDeviceToHost));
    CudaSafeCall(hipMemcpy(max_p, g_max_o, 2*sizeof(float), hipMemcpyDeviceToHost));
    CudaSafeCall(hipMemcpy(max_idx_p, g_max_idx_o, 2*sizeof(int), hipMemcpyDeviceToHost));
    hipDeviceSynchronize();
    *nargmax1_p = max_idx_p[0];
    *nargmax2_p = max_idx_p[1];
    *maxr_p = max_p[0];
    *maxi_p = max_p[1];
    hipFree(dev_ker);
    hipFree(dev_res);
    hipFree(g_nscore_i);
    hipFree(g_max_i);
    hipFree(g_max_idx_i);
    hipFree(g_nscore_o);
    hipFree(g_max_o);
    hipFree(g_max_idx_o);
    return 0;
}