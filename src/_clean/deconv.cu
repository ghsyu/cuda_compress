#include "hip/hip_runtime.h"
/*
 * Some additional deconvolution functions for AIPY, written in C++ and CUDA.  These are
 * mostly for speed-critical applications. 
 *
 * Author: Aaron Parsons, Gilbert Hsyu
 */

#include "deconv.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

__global__ void sum_max(float *g_nscore, float *g_max, int* g_max_idx){
    extern __shared__ float s_nscore[];
    float *s_max = s_nscore + blockDim.x;
    int *s_max_idx =(int *) s_nscore + 3*blockDim.x;
    int tid = threadIdx.x;
    int i = blockIdx.x*blockDim.x + tid;
    for (unsigned int s=blockDim.x/2; s>0; s>>=1){
        if (tid < s){
            s_nscore[tid] += s_nscore[tid + s];
            if ((s_max[2*tid]*s_max[2*tid] + s_max[2*tid+1]*s_max[2*tid+1]) < \
                (s_max[2*(tid+s)]*s_max[2*(tid+s)] + s_max[2*(tid+s)+1]*s_max[2*(tid+s)+1])){
                s_max[2*tid]         = s_max[2*(tid+s)];
                s_max[2*tid+1]       = s_max[2*(tid+s)+1];
                s_max_idx[2*tid]     = s_max_idx[2*(tid+s)];
                s_max_idx[2*tid + 1] = s_max_idx[2*(tid+s)+1];
            }
        }
        __syncthreads();
    }
    if (threadIdx.x == 0 && threadIdx.y == 0){
        g_nscore[i]        = s_nscore[0];
        g_max[2*i]         = s_max[0];
        g_max[2*i+1]       = s_max[1];
        g_max_idx[2*(i)]   = s_max_idx[0];
        g_max_idx[2*(i)+1] = s_max_idx[1];
    }
}

__global__ void clean2dc(int *dim1_p, int *dim2_p, int *argmax1_p, int *argmax2_p, float *stepr_p, \
                        float *stepi_p, float *ker, float *res, float *val_arr, float *g_nscore, \
                        float *g_max, int *g_max_idx){
    int dim1 = *dim1_p;
    int dim2 = *dim2_p;
    int argmax1 = *argmax1_p;
    int argmax2 = *argmax2_p;
    float stepr = *stepr_p;
    float stepi = *stepi_p;
    float valr, vali;
    extern __shared__ float s_nscore[];
    float *s_max   = s_nscore + blockDim.x*blockDim.y;
    int *s_max_idx = (int *)s_nscore + 3*blockDim.x*blockDim.y;
    int n1 = threadIdx.x + blockIdx.x * blockDim.x;
    int n2 = threadIdx.y + blockIdx.y * blockDim.y;
    int tid = n1 + blockDim.x*n2;
    s_max_idx[2*tid] = n1;
    s_max_idx[2*tid+1] = n2;
    if ((n1 < dim1) && (n2 < dim2)){
        int wrap_n1 = (n1 + argmax1) % dim1;
        int wrap_n2 = (n2 + argmax2) % dim2;
        res[2*(wrap_n1 + wrap_n2*dim1)]     -= (ker[2*(n1 + n2*dim1)] * stepr - ker[2*(n1 + n2*dim1)+1] * stepi);
        res[2*(wrap_n1 + wrap_n2*dim1) + 1] -= (ker[2*(n1 + n2*dim1)] * stepi + ker[2*(n1 + n2*dim1)+1] * stepr);
        valr = res[2*(wrap_n1 + wrap_n2*dim1)];
        vali = res[2*(wrap_n1 + wrap_n2*dim1) + 1];
        s_nscore[threadIdx.x + threadIdx.y*blockDim.x] = valr*valr+vali*vali;
        s_max[2*(threadIdx.x + threadIdx.y*blockDim.x)] = valr;
        s_max[2*(threadIdx.x + threadIdx.y*blockDim.x)+1] = vali;
    } else {
        s_nscore[tid] = 0;
        s_max[tid] = -1;
    }
    for (unsigned int s=blockDim.x/2; s>0; s>>=1){
        if (tid < s){
            s_nscore[tid] += s_nscore[tid + s];
            if ((s_max[2*tid]*s_max[2*tid] + s_max[2*tid+1]*s_max[2*tid+1]) < \
                (s_max[2*(tid+s)]*s_max[2*(tid+s)] + s_max[2*(tid+s)+1]*s_max[2*(tid+s)+1])){
                s_max[2*tid]         = s_max[2*(tid+s)];
                s_max[2*tid+1]       = s_max[2*(tid+s)+1];
                s_max_idx[2*tid]     = s_max_idx[2*(tid+s)];
                s_max_idx[2*tid+1]   = s_max_idx[2*(tid+s)+1];
            }
        }
        __syncthreads();
    }
    if (threadIdx.x == 0 && threadIdx.y == 0){
        g_nscore[blockIdx.x+blockDim.x*blockIdx.y]        = s_nscore[0];
        g_max[2*(blockIdx.x+blockDim.x*blockIdx.y)]       = s_max[0];
        g_max[2*(blockIdx.x+blockDim.x*blockIdx.y)+1]     = s_max[1];
        g_max_idx[2*(blockIdx.x+blockDim.x*blockIdx.y)]   = s_max_idx[0];
        g_max_idx[2*(blockIdx.x+blockDim.x*blockIdx.y)+1] = s_max_idx[1];
    }
    return;
}

//   ____ _                  ____     _      
//  / ___| | ___  __ _ _ __ |___ \ __| | ___ 
// | |   | |/ _ \/ _` | '_ \  __) / _` |/ __|
// | |___| |  __/ (_| | | | |/ __/ (_| | (__ 
//  \____|_|\___|\__,_|_| |_|_____\__,_|\___|  
// Does a 2d complex-valued clean
float *clean_2d_c_GPU(float *res, float *ker,
        double gain, int maxiter, \
        int stop_if_div, \
        float *stepr_p, float *stepi_p, \
        int ker_len, int res_len, int dim1, int dim2,
        float *nscore_p, float *maxr_p, float *maxi_p, int *argmax1_p, int *argmax2_p) {
    int gridx, gridy;
    float *dev_ker, *dev_res, *dev_stepr, *dev_stepi, *dev_val_arr, *g_nscore, \
        *g_max;
    int *dev_argmax1, *dev_argmax2,
        *dev_dim1, *dev_dim2, *g_max_idx;
    float stepr = *stepr_p;
    float stepi = *stepi_p;
    int argmax1 = *argmax1_p;
    int argmax2 = *argmax2_p;
    float max_p[2];
    int max_idx_p[2];
    //Ceiling division of dim1/BLOCKSIZEX and dim2/BLOCKSIZEY
    gridx = (dim1 % BLOCKSIZEX == 0) ? dim1/BLOCKSIZEX : dim1/BLOCKSIZEX + 1;
    gridy = (dim2 % BLOCKSIZEY == 0) ? dim2/BLOCKSIZEY : dim2/BLOCKSIZEY + 1;
    dim3 grid(gridx, gridy);
    dim3 blocksize(BLOCKSIZEX, BLOCKSIZEY);
    
    CudaSafeCall(hipMalloc((void**) &dev_ker,      ker_len));
    CudaSafeCall(hipMalloc((void**) &dev_res,      res_len));
    CudaSafeCall(hipMalloc((void**) &dev_dim1,     sizeof(int)));
    CudaSafeCall(hipMalloc((void**) &dev_dim2,     sizeof(int)));
    CudaSafeCall(hipMalloc((void**) &dev_argmax1,  sizeof(int)));
    CudaSafeCall(hipMalloc((void**) &dev_argmax2,  sizeof(int)));
    CudaSafeCall(hipMalloc((void**) &dev_stepr,    sizeof(float)));
    CudaSafeCall(hipMalloc((void**) &dev_stepi,    sizeof(float)));
    CudaSafeCall(hipMalloc((void**) &dev_val_arr,  2*sizeof(float)*dim1*dim2));
    CudaSafeCall(hipMalloc((void**) &g_nscore,     sizeof(float)*gridx*gridy));
    CudaSafeCall(hipMalloc((void**) &g_max,        2*sizeof(float)*gridx*gridy));
    CudaSafeCall(hipMalloc((void**) &g_max_idx,    2*sizeof(int)*gridx*gridy));
    
    CudaSafeCall(hipMemcpy(dev_ker,      ker,      ker_len,       hipMemcpyHostToDevice));
    CudaSafeCall(hipMemcpy(dev_res,      res,      res_len,       hipMemcpyHostToDevice));
    CudaSafeCall(hipMemcpy(dev_dim1,     &dim1,    sizeof(int),   hipMemcpyHostToDevice));
    CudaSafeCall(hipMemcpy(dev_dim2,     &dim2,    sizeof(int),   hipMemcpyHostToDevice));
    CudaSafeCall(hipMemcpy(dev_argmax1,  &argmax1, sizeof(int),   hipMemcpyHostToDevice));
    CudaSafeCall(hipMemcpy(dev_argmax2,  &argmax2, sizeof(int),   hipMemcpyHostToDevice));
    CudaSafeCall(hipMemcpy(dev_stepr,    &stepr,   sizeof(float), hipMemcpyHostToDevice));
    CudaSafeCall(hipMemcpy(dev_stepi,    &stepi,   sizeof(float), hipMemcpyHostToDevice));
    
    int smemsize = 5*BLOCKSIZEX*BLOCKSIZEY*sizeof(float);
    // Take next step and compute score
    clean2dc<<<grid, blocksize, smemsize>>>(dev_dim1, dev_dim2, dev_argmax1, dev_argmax2, dev_stepr, \
                                dev_stepi, dev_ker, dev_res, dev_val_arr, g_nscore, g_max, g_max_idx);
    CudaSafeCall(hipMemcpy(res, dev_res, res_len, hipMemcpyDeviceToHost));
    CudaCheckError();
    
    int bsize = BLOCKSIZEX*BLOCKSIZEY;
    int gridsize = gridx*gridy/bsize;
    if ((gridsize % bsize) != 0 || gridsize == 0){
        gridsize += 1;
    }
    while(1){
        dim3 grid(gridsize, 1, 1);
        sum_max<<<grid, bsize, smemsize>>>(g_nscore, g_max, g_max_idx);
        CudaCheckError();
        if (gridsize % bsize == 0 && gridsize/bsize > 1){
            gridsize = gridsize/bsize;
        } else if (gridsize/bsize > 1){
            gridsize = gridsize/bsize +1;
        } else {
            dim3 grid(1,1,1);
            break;
        }
    }
    CudaCheckError();
    CudaSafeCall(hipMemcpy(nscore_p, g_nscore, sizeof(float), hipMemcpyDeviceToHost));
    CudaSafeCall(hipMemcpy(max_p, g_max, 2*sizeof(float), hipMemcpyDeviceToHost));
    CudaSafeCall(hipMemcpy(max_idx_p, g_max_idx, 2*sizeof(int), hipMemcpyDeviceToHost));
    *argmax1_p = max_idx_p[0];
    *argmax2_p = max_idx_p[1];
    *maxr_p = max_p[0];
    *maxi_p = max_p[1];
    hipFree(dev_ker);
    hipFree(dev_res);
    hipFree(dev_dim1);
    hipFree(dev_dim2);
    hipFree(dev_argmax1);
    hipFree(dev_argmax2);
    hipFree(dev_stepr);
    hipFree(dev_stepi);
    hipFree(dev_val_arr);
    hipFree(g_nscore);
    hipFree(g_max);
    hipFree(g_max_idx);
    return 0;
}